#include "hip/hip_runtime.h"
#include <iostream>
#include <TNL/Devices/Host.h>
#include <TNL/Containers/Vector.h>


using namespace TNL;
using namespace TNL::Containers;

int
main( int argc, char* argv[] )
{
   const int size = 11;

   TNL::Containers::Vector< float, TNL::Devices::Cuda > a( size ), b( size ), c( size );

   a.forAllElements(
      [] __cuda_callable__( int i, float& value )
      {
         value = 3.14 * ( i - 5.0 ) / 5.0;
      } );

   b = a * a;
   c = 3 * a + sign( a ) * sin( a );
   std::cout << "a = " << a << std::endl;
   std::cout << "sin( a ) = " << sin( a ) << std::endl;
   std::cout << "abs( sin( a ) ) = " << abs( sin( a ) ) << std::endl;
   std::cout << "b = " << b << std::endl;
   std::cout << "c = " << c << std::endl;
}
